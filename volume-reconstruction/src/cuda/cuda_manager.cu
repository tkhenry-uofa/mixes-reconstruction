#include <iostream>
#include <stdexcept>
#include <chrono>

#include "delay_and_sum_kernel.cuh"
#include "cuda_manager.cuh"

CudaManager::CudaManager( defs::TxConfig config) : _tx_config(config), _textures({0,0,0}), _position_arrays(), _vox_counts({0,0,0,0}), _rf_dims({0,0,0}), _d_volume(nullptr)
{
	std::cout << "Allocating GPU Memory" << std::endl;
	hipError_t cuda_status = hipSetDevice(0);

	_position_arrays[0] = nullptr;
	_position_arrays[1] = nullptr;
	_position_arrays[2] = nullptr;

	if (cuda_status != hipSuccess)
	{
		std::cerr << "Failed set cuda device" << std::endl;
		throw std::invalid_argument("Invalid cuda device");
	}
}

CudaManager::~CudaManager()
{
	cleanupMemory();
}

void
CudaManager::cleanupMemory()
{
	hipFree(_d_rf_data);
	hipFree(_d_loc_data);
	hipFree(_d_volume);

	hipFreeArray(_position_arrays[0]);
	hipFreeArray(_position_arrays[1]);
	hipFreeArray(_position_arrays[2]);
}

bool
CudaManager::configureVolume(const defs::VolumeDims& dims)
{
	std::vector<float> x_range;
	std::vector<float> y_range;
	std::vector<float> z_range;

	// Destroy old data
	hipDestroyTextureObject(_textures.x);
	hipDestroyTextureObject(_textures.y);
	hipDestroyTextureObject(_textures.z);

	hipFreeArray(_position_arrays[0]);
	hipFreeArray(_position_arrays[1]);
	hipFreeArray(_position_arrays[2]);

	hipFree(_d_volume);

	for (float x = dims.x_min; x <= dims.x_max; x += dims.resolution) {
		x_range.push_back(x);
	}
	for (float y = dims.y_min; y <= dims.y_max; y += dims.resolution) {
		y_range.push_back(y);
	}
	for (float z = dims.z_min; z <= dims.z_max; z += dims.resolution) {
		z_range.push_back(z);
	}

	_vox_counts = { z_range.size(), y_range.size(), z_range.size(), x_range.size() * y_range.size() * z_range.size() };


	hipError_t cuda_status = hipMalloc((void**)&_d_volume, _vox_counts.w * sizeof(float));
	RETURN_IF_ERROR(cuda_status, "Failed to malloc volume on device.")

	// TEXTURE SETUP
	// 32 bits in the channel 
	hipChannelFormatDesc channel_desc = hipCreateChannelDesc(sizeof(float) * 8, 0, 0, 0, hipChannelFormatKindFloat);

	hipTextureDesc tex_desc;
	memset(&tex_desc, 0, sizeof(hipTextureDesc));
	tex_desc.addressMode[0] = hipAddressModeClamp;
	tex_desc.filterMode = hipFilterModePoint;
	tex_desc.readMode = hipReadModeElementType;
	tex_desc.normalizedCoords = false;

	hipResourceDesc tex_res_desc;
	memset(&tex_res_desc, 0, sizeof(hipResourceDesc));
	tex_res_desc.resType = hipResourceTypeArray;
	
	hipError_t malloc_status = hipMallocArray(&_position_arrays[0], &channel_desc, _vox_counts.x);
	hipError_t memcpy_status = hipMemcpyToArray(_position_arrays[0], 0, 0, x_range.data(), _vox_counts.x * sizeof(float), hipMemcpyHostToDevice);
	tex_res_desc.res.array.array = _position_arrays[0];
	hipError_t bind_status = hipCreateTextureObject(&_textures.x, &tex_res_desc, &tex_desc, NULL);

	if (malloc_status != hipSuccess || memcpy_status != hipSuccess || bind_status != hipSuccess)
	{
		std::cerr << "Failed to create x texture." << std::endl;
		return false;
	}
	
	malloc_status = hipMallocArray(&_position_arrays[1], &channel_desc, _vox_counts.y);
	memcpy_status = hipMemcpyToArray(_position_arrays[1], 0, 0, y_range.data(), _vox_counts.y * sizeof(float), hipMemcpyHostToDevice);
	tex_res_desc.res.array.array = _position_arrays[1];
	bind_status = hipCreateTextureObject(&_textures.y, &tex_res_desc, &tex_desc, NULL);

	if (malloc_status != hipSuccess || memcpy_status != hipSuccess || bind_status != hipSuccess)
	{
		std::cerr << "Failed to create y texture." << std::endl;
		return false;
	}

	
	malloc_status = hipMallocArray(&_position_arrays[2], &channel_desc, _vox_counts.z);
	memcpy_status = hipMemcpyToArray(_position_arrays[2], 0, 0, z_range.data(), _vox_counts.z * sizeof(float), hipMemcpyHostToDevice);
	tex_res_desc.res.array.array = _position_arrays[2];
	bind_status = hipCreateTextureObject(&_textures.z, &tex_res_desc, &tex_desc, NULL);

	if (malloc_status != hipSuccess || memcpy_status != hipSuccess || bind_status != hipSuccess)
	{
		std::cerr << "Failed to create z texture." << std::endl;
		return false;
	}
	
	return true;
}

bool
CudaManager::transferLocData(const std::vector<float>& loc_data)
{

	if (_d_loc_data != nullptr)
	{
		hipFree(_d_loc_data);
	}

	hipError_t cuda_status = hipMalloc((void**)&_d_loc_data, loc_data.size() * sizeof(float));
	RETURN_IF_ERROR(cuda_status, "Failed to malloc location array on device.")

	cuda_status = hipMemcpy(_d_loc_data, (void*)loc_data.data(), loc_data.size() * sizeof(float), hipMemcpyHostToDevice);
	RETURN_IF_ERROR(cuda_status, "Failed to copy location array to device.")

	return cuda_status == hipSuccess;
}

bool
CudaManager::transferRfData(const std::vector<std::complex<float>>& rf_data, const defs::RfDataDims& rf_dims)
{
	if (_d_rf_data != nullptr)
	{
		hipFree(_d_rf_data);
	}

	_rf_dims = rf_dims;
	size_t rf_data_size= rf_data.size();
	hipError_t cuda_status = hipMalloc((void**)&_d_rf_data, rf_data_size * sizeof(std::complex<float>));
	RETURN_IF_ERROR(cuda_status, "Failed to malloc rf data on device.")

	cuda_status = hipMemcpy(_d_rf_data, (void*)rf_data.data(), rf_data.size() * sizeof(std::complex<float>), hipMemcpyHostToDevice);
	RETURN_IF_ERROR(cuda_status, "Failed to copy rf data to device.")

	return cuda_status == hipSuccess;
}

bool
CudaManager::beamform(std::vector<float>** volume)
{
	*volume = nullptr;
	defs::KernelConstants const_struct =
	{
		_rf_dims.element_count,
		_rf_dims.sample_count,
		_tx_config.src_location,
		_rf_dims.tx_count,
		_tx_config.transmit_type,
		_vox_counts
	};

	hipError_t cuda_status = helpers::copy_constants(const_struct);
	RETURN_IF_ERROR(cuda_status, "Failed to copy constants to device.")


	dim3 gridDim((unsigned int)_vox_counts.x, (unsigned int)_vox_counts.y, (unsigned int)_vox_counts.z);
	auto start = std::chrono::high_resolution_clock::now();
	kernels::complexDelayAndSum <<<gridDim, THREADS_PER_BLOCK >>> (_d_rf_data, _d_loc_data, _d_volume, _textures);

	cuda_status = hipGetLastError();
	RETURN_IF_ERROR(cuda_status, "Kernel failed.")
	cuda_status = hipDeviceSynchronize();
	RETURN_IF_ERROR(cuda_status, "Sync failed.")

	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed = end - start;
	std::cout << "Kernel duration: " << elapsed.count() << " seconds" << std::endl;

	*volume = new std::vector<float>(_vox_counts.w);
	cuda_status = hipMemcpy((*volume)->data(), _d_volume, _vox_counts.w * sizeof(float), hipMemcpyDeviceToHost);
	RETURN_IF_ERROR(cuda_status, "Copying volume to CPU failed.")

	return cuda_status == hipSuccess;
	
}